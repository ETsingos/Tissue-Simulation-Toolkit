#include "hip/hip_runtime.h"
#define ARRAY_SIZE 33

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)

void PDE::InitialiseCuda(CellularPotts *cpm){
  cout << "Start cuda init" << endl;
  //AllocateTridiagonalvars(sizex, sizey);

  hipMalloc((void**) &d_couplingcoefficient, sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMalloc((void**) &d_celltype, sizex*sizey*sizeof(int));
  hipMalloc((void**) &d_sigmafield, sizex*sizey*sizeof(int));

  hipMalloc((void**) &d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMemcpy(d_PDEvars, PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);
  hipMalloc((void**) &d_alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMemcpy(d_alt_PDEvars, alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);


  //Needed for ADI steps
  gpuErrchk(hipMallocManaged(&upperH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&diagH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&lowerH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&BH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&upperV, sizey*sizex*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&diagV, sizey*sizex*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&lowerV, sizey*sizex*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&BV, sizey*sizex*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&next_stepsize, sizey*sizex*sizeof(PDEFIELD_TYPE)));

  handleH = 0;
  pbuffersizeH = 0;
  pbufferH = NULL;
  statusH=hipsparseCreate(&handleH);
  #ifdef PDEFIELD_DOUBLE
    hipsparseDgtsvInterleavedBatch_bufferSizeExt(handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey, &pbuffersizeH); //Compute required buffersize for horizontal sweep
  #else
    hipsparseSgtsvInterleavedBatch_bufferSizeExt(handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey, &pbuffersizeH); //Compute required buffersize for horizontal sweep
  #endif
  gpuErrchk(hipMalloc( &pbufferH, sizeof(char)* pbuffersizeH));
  

  handleV = 0;
  pbuffersizeV = 0;
  pbufferV = NULL;
  statusV=hipsparseCreate(&handleV);
  #ifdef PDEFIELD_DOUBLE
    hipsparseDgtsvInterleavedBatch_bufferSizeExt(handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex, &pbuffersizeV); //Compute required buffersize for vertical sweep
  #else
    hipsparseSgtsvInterleavedBatch_bufferSizeExt(handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex, &pbuffersizeV); //Compute required buffersize for vertical sweep
  #endif
  gpuErrchk(hipMalloc( &pbufferV, sizeof(char)* pbuffersizeV));

  cout << "End cuda init" << endl;
}


__global__ void InitializeDiagonals(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* lowerH, PDEFIELD_TYPE* upperH, PDEFIELD_TYPE* diagH, PDEFIELD_TYPE* lowerV, PDEFIELD_TYPE* upperV, PDEFIELD_TYPE* diagV, PDEFIELD_TYPE* couplingcoefficient){
  //This function could in theory be parellelized further, split into 6 (each part only assigning 1 value.), but this is probably slower
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc; //position we currently want to assign to
  int yloc;
  int idcc; //id corresponding to the couplingcoefficient (+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id/sizey; //needed to obtain interleaved format
    yloc = id%sizey;
    idcc = xloc*sizey + yloc;
    if(xloc == 0){
      lowerH[id] = 0;
      diagH[id] = couplingcoefficient[idcc+sizey]/dx2 + twooverdt;
      upperH[id] = -couplingcoefficient[idcc+sizey]/dx2;  
    }
    else if(xloc == sizex -1){
      lowerH[id] = -couplingcoefficient[idcc-sizey]/dx2;
      diagH[id] = couplingcoefficient[idcc-sizey]/dx2 + twooverdt;
      upperH[id] = 0;
    }
    else{
      lowerH[id] = -couplingcoefficient[idcc-sizey]/dx2;
      diagH[id] = (couplingcoefficient[idcc+sizey]+couplingcoefficient[idcc-sizey])/dx2 + twooverdt;
      upperH[id] = -couplingcoefficient[idcc+sizey]/dx2;
    }

    xloc = id%sizex; //needed to obtain interleaved format
    yloc = id/sizex;
    idcc = xloc*sizey + yloc;
    if(yloc == 0){
      lowerV[id] = 0;
      diagV[id] = couplingcoefficient[idcc+1]/dx2 + twooverdt;
      upperV[id] = -couplingcoefficient[idcc+1]/dx2;
    }
    else if(yloc == sizey -1){
      lowerV[id] = -couplingcoefficient[idcc-1]/dx2;
      diagV[id] = couplingcoefficient[idcc-1]/dx2 + twooverdt;
      upperV[id] = 0;
    }
    else{
      lowerV[id] = -couplingcoefficient[idcc-1]/dx2;
      diagV[id] = (couplingcoefficient[idcc+1]+couplingcoefficient[idcc-1])/dx2 + twooverdt;
      upperV[id] = -couplingcoefficient[idcc+1]/dx2;
      
    }
  }
}

__global__ void InitializeHorizontalVectors(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* couplingcoefficient, PDEFIELD_TYPE* alt_PDEvars){
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc;
  int yloc;
  int idcc; //id corresponding to the couplingcoefficient and alt_PDEvars(+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id/sizey; //needed to obtain interleaved format
    yloc = id%sizey;
    idcc = xloc*sizey + yloc;

    if (yloc == 0)
      BH[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+1]*(alt_PDEvars[idcc+1] - alt_PDEvars[idcc]))/dx2; 
    else if (yloc == sizey-1)
      BH[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc-1]*(alt_PDEvars[idcc-1] - alt_PDEvars[idcc]))/dx2;  
    else 
      BH[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+1]*(alt_PDEvars[idcc+1] - alt_PDEvars[idcc]) + couplingcoefficient[idcc-1]*(alt_PDEvars[idcc-1] - alt_PDEvars[idcc]))/dx2;
  }
}

__global__ void InitializeVerticalVectors(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* BV, PDEFIELD_TYPE* couplingcoefficient, PDEFIELD_TYPE* alt_PDEvars){

  //This function could in theory be parellelized further, split into 6 (each part only assigning 1 value.), but this is probably slower
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc;
  int yloc;
  int idcc; //id corresponding to the couplingcoefficient and alt_PDEvars(+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id%sizex;
    yloc = id/sizex;
    idcc = xloc*sizey + yloc;

    if (xloc == 0)
      BV[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+sizey]*(alt_PDEvars[idcc+sizey] - alt_PDEvars[idcc]))/dx2; 
    else if (xloc == sizex-1)
      BV[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc-sizey]*(alt_PDEvars[idcc-sizey] - alt_PDEvars[idcc]))/dx2;  
    else 
      BV[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+sizey]*(alt_PDEvars[idcc+sizey] - alt_PDEvars[idcc]) + couplingcoefficient[idcc-sizey]*(alt_PDEvars[idcc-sizey] - alt_PDEvars[idcc]))/dx2;
  }

}



__global__ void NewPDEfieldH0(int sizex, int sizey, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* PDEvars){ //Take the values from BH and assign the new values of the first layers of PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride)
    PDEvars[id] = BH[id];      
}


__global__ void NewPDEfieldV0(int sizex, int sizey, PDEFIELD_TYPE* BV, PDEFIELD_TYPE* PDEvars){ //Take the values from BV and assign the new values of the first layers of PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride){
    PDEvars[sizey*(id%sizex)+id/sizex] = BV[id]; //Conversion is needed because PDEvars iterates over columns first and then rows, while BV does the opposite 
  }
}


__global__ void NewPDEfieldOthers(int sizex, int sizey, int layers, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars){ //copy the other values from alt_PDEvars to PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index+sizex*sizey; id < layers*sizex*sizey; id += stride)
    PDEvars[id] = alt_PDEvars[id]; 
}

__device__ void derivsFitzHughNagumo(PDEFIELD_TYPE current_time, PDEFIELD_TYPE* y, PDEFIELD_TYPE* dydt, bool celltype2, int* sigmafield, PDEFIELD_TYPE pacing_interval, PDEFIELD_TYPE pacing_duration, PDEFIELD_TYPE pacing_strength, int id, PDEFIELD_TYPE interval_beats, PDEFIELD_TYPE pulse_duration, PDEFIELD_TYPE pulse_strength,  PDEFIELD_TYPE a, PDEFIELD_TYPE b, PDEFIELD_TYPE tau, PDEFIELD_TYPE* FHN_a, PDEFIELD_TYPE* FHN_b, PDEFIELD_TYPE* FHN_tau ){
  
    a = 0.1;
    PDEFIELD_TYPE epsilon = 10;
    PDEFIELD_TYPE beta = -1.0;
    PDEFIELD_TYPE RIext = 0;
    PDEFIELD_TYPE c = 0.191;
    PDEFIELD_TYPE timescale = 40;
  
  int sigma = sigmafield[id];
  if (fmod(current_time, interval_beats) < pulse_duration && celltype2)
    RIext = pulse_strength;


  dydt[0] = timescale*(epsilon*(y[0]*(1-y[0])*(y[0]-beta)) - y[1] + RIext);
  dydt[1] = timescale*((y[0] -a*y[1]+c));      

}

__global__ void ODEstepFE(PDEFIELD_TYPE dt, PDEFIELD_TYPE ddt, double thetime, int layers, int sizex, int sizey, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars, int* celltype, int* sigmafield, PDEFIELD_TYPE* next_stepsize, PDEFIELD_TYPE stepsize_min, PDEFIELD_TYPE pacing_interval, PDEFIELD_TYPE I_Na_factor, PDEFIELD_TYPE I_f_factor, PDEFIELD_TYPE I_Kr_factor){
  //PDEFIELD_TYPE ddt = 2e-7; //for couplingcoefficient 1e-4 
  //PDEFIELD_TYPE ddt = 1e-6; //for couplingcoefficient 1e-5
  int nr_of_iterations = round(dt/ddt);
  if (fabs(dt/ddt - nr_of_iterations) > 0.001)
    printf("dt and ddt do not divide properly!");
  PDEFIELD_TYPE begin_time,stepsize_next,stepsize_did,stepsize, end_time;
  PDEFIELD_TYPE yscal[ARRAY_SIZE];
  PDEFIELD_TYPE y[ARRAY_SIZE];
  PDEFIELD_TYPE y_new[ARRAY_SIZE];
  PDEFIELD_TYPE dydt[ARRAY_SIZE];
  PDEFIELD_TYPE current_time;
  PDEFIELD_TYPE MaxTimeError = 5e-7;
  PDEFIELD_TYPE stepsize_overshot;
  bool overshot = false;
  bool celltype2 = false;
  int i;

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride){
    if (celltype[id] < 1){
      for (i = 0; i < layers; i++) //fill with current PDE values
        alt_PDEvars[i*sizex*sizey + id]= PDEvars[i*sizex*sizey + id];
    }
    
    else{
      celltype2 = false; 
      if (celltype[id] == 2)
        celltype2 = true;
      begin_time = thetime;
      current_time = thetime;
      end_time = thetime + dt;
      stepsize=next_stepsize[id];
      for (i=0;i<layers;i++)
        y[i]=PDEvars[i*sizex*sizey + id];
      for (int it = 0; it < nr_of_iterations; it++){

        overshot = false;
        if (celltype2)
          derivsFabbriSeveri(current_time,y,dydt,pacing_interval, I_f_factor, I_Kr_factor, id);
        else{
          derivsMaleckar(current_time,y,dydt,pacing_interval, I_Na_factor, id, 0);
        }
        //derivsFitzHughNagumo(current_time,y,dydt,celltype2, sigmafield, pacing_interval,pacing_duration,pacing_strength, id, FHN_interval_beats, FHN_pulse_duration, FHN_pulse_strength,  a, b, tau, FHN_a, FHN_b, FHN_tau);
        current_time += ddt;
        if (it == nr_of_iterations-1) { //Are we done?
          for (i=0;i<layers;i++) {
            alt_PDEvars[i*sizex*sizey + id] = y[i]+ddt*dydt[i];
          }
        }
        else{  
          for (i=0;i<layers;i++) {
            y[i]=y[i]+ddt*dydt[i];  
          }
        }
      }
    }
  }
}



__global__ void CopyAltToOriginalPDEvars(int sizex, int sizey, int layers, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < layers*sizex*sizey; id += stride){
    PDEvars[id] = alt_PDEvars[id]; 
  }
}

__global__ void CopyOriginalToAltPDEvars(int sizex, int sizey, int layers, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < layers*sizex*sizey; id += stride){
    alt_PDEvars[id] = PDEvars[id]; 
  }
}

void PDE::cuPDEsteps(CellularPotts * cpm, int repeat){
  if (thetime == 0 && par.SF_all)
    InitializeSFComputation(cpm);
  //copy current couplingcoefficient matrix and celltype matrix from host to device
  couplingcoefficient = cpm->getCouplingCoefficient();
  //couplingcoefficient = cpm->getCouplingCoefficient_Gradient();

  //int** cellnumber = cpm -> getSigma(); 
  hipError_t errSync;
  hipError_t errAsync;
  celltype = cpm->getTau();
  sigmafield = cpm->getSigma(); 
  hipMemcpy(d_couplingcoefficient, couplingcoefficient[0], sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice); 
  hipMemcpy(d_celltype, celltype[0], sizex*sizey*sizeof(int), hipMemcpyHostToDevice); 
  hipMemcpy(d_sigmafield, sigmafield[0], sizex*sizey*sizeof(int), hipMemcpyHostToDevice); 
  hipMemcpy(d_PDEvars, PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice); 

  
  int nr_blocks = sizex*sizey/par.threads_per_core + 1;
  PDEFIELD_TYPE Cm_maleckar = 50; //in nF
  PDEFIELD_TYPE I_m;
  bool afterdiffusion;

  for (int iteration = 0; iteration < repeat; iteration++){
    if (par.SF_all){
      cuSFChecker();
    }
      //cout << "Iteration = " << iteration << endl;

      //setup matrices for upperdiagonal, diagonal and lower diagonal for both the horizontal and vertical direction, since these remain the same during once MCS
    InitializeDiagonals<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, lowerH, upperH, diagH, lowerV, upperV, diagV, d_couplingcoefficient);
    hipDeviceSynchronize();
    errSync  = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
      printf("Async kernel error: %s\n", hipGetErrorString(errAsync));


    cuODEstep();
    afterdiffusion = false;
    if (par.SF_all){
      cuCopyVoltageForSF(afterdiffusion);
    }
  
    cuHorizontalADIstep();
    afterdiffusion = true;
    if (par.SF_all){
      cuCopyVoltageForSF(afterdiffusion);
    }

    //increase time by dt/2
    thetime = thetime + dt/2;  
    cuODEstep();
    afterdiffusion = false;
    if (par.SF_all){
      cuCopyVoltageForSF(afterdiffusion);
      cout << "This shouldn't happen!";
    }


    cuVerticalADIstep();
    afterdiffusion = true;
    if (par.SF_all){
      cuCopyVoltageForSF(afterdiffusion);
      cout << "This shouldn't happen!";
    }
      
    //hipMemcpy(alt_PDEvars, d_alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
    //cout << "After second FE step, alt_PDEvars[23885] = " << alt_PDEvars[23885] << endl;
    if (par.SF_all){
      cout << "This shouldn't happen!";
      if (SF_start_one && !SF_end_one && par.SF_one_pixel)
        cuComputeSFOne();
      
      if (SF_in_progress && !SF_all_done && par.SF_all)
        cuWriteSFData();
    }
    

    
    //increase time by dt/2
    thetime = thetime + dt/2; 

    if (par.activation_times){    
      CheckActivationTimes<<<par.number_of_cores, par.threads_per_core>>>(thetime, d_Activation_times_array, d_sigmafield,  d_PDEvars, sizex, sizey);
    }   
  }
  hipMemcpy(PDEvars, d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  cuPDEVarsToFiles();
  if (par.activation_times){ 
    hipMemcpy(Activation_times_array, d_Activation_times_array, sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);   
    cuWriteActivationTimes();
  }
}

void PDE::cuODEstep(){
  //Do an ODE step of size dt/2
  hipError_t errSync;
  hipError_t errAsync;
  //ODEstepRL_Paci<<<nr_blocks, par.threads_per_core>>>(dt/2, ddt, thetime, layers, sizex, sizey, d_PDEvars, d_alt_PDEvars, d_celltype, next_stepsize, min_stepsize, par.eps, pacing_interval, par.pacing_duration, par.pacing_strength);
  //ODEstepRKA<<<par.number_of_cores, par.threads_per_core>>>(dt/2, thetime, layers, sizex, sizey, d_PDEvars, d_alt_PDEvars, d_celltype, next_stepsize, min_stepsize, par.eps, pacing_interval, par.pacing_duration, par.pacing_strength);
  ODEstepFE<<<par.number_of_cores, par.threads_per_core>>>(dt/2, ddt, thetime, layers, sizex, sizey, d_PDEvars, d_alt_PDEvars, d_celltype, d_sigmafield, next_stepsize, min_stepsize, pacing_interval, par.I_f_factor, par.I_Kr_factor, par.I_Na_factor);
  //CopyOriginalToAltPDEvars<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, layers, d_PDEvars, d_alt_PDEvars);

  //hipMemcpy(alt_PDEvars, d_alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
  //cout << "After second FE step, alt_PDEvars[4305] = " << alt_PDEvars[4305] << endl;
  cuErrorChecker(errSync, errAsync);
}

void PDE::cuHorizontalADIstep(){
  //Do a horizontal ADI sweep of size dt/2
  hipError_t errSync;
  hipError_t errAsync;
  InitializeHorizontalVectors<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, BH, d_couplingcoefficient, d_alt_PDEvars);
  cuErrorChecker(errSync, errAsync);
  #ifdef PDEFIELD_DOUBLE
    statusH = hipsparseDgtsvInterleavedBatch(handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey, pbufferH);
  #else
    statusH = hipsparseSgtsvInterleavedBatch(handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey, pbufferH);
  #endif
  if (statusH != HIPSPARSE_STATUS_SUCCESS)
  {
    cout << statusH << endl;
  }
  NewPDEfieldH0<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, BH, d_PDEvars);    
  cuErrorChecker(errSync, errAsync);
  NewPDEfieldOthers<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, layers, BV, d_PDEvars, d_alt_PDEvars); //////
  cuErrorChecker(errSync, errAsync);

  //hipMemcpy(PDEvars, d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
  //cout << "After second FE step, PDEvars[4305] = " << PDEvars[4305] << endl;

}

void PDE::cuVerticalADIstep(){
  //Do a vertical ADI sweep of size dt/2
  hipError_t errSync;
  hipError_t errAsync;
  InitializeVerticalVectors<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, BV, d_couplingcoefficient, d_alt_PDEvars);
  cuErrorChecker(errSync, errAsync);
  #ifdef PDEFIELD_DOUBLE
    statusV = hipsparseDgtsvInterleavedBatch(handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex, pbufferV);
  #else
    statusV = hipsparseSgtsvInterleavedBatch(handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex, pbufferV);
  #endif
  if (statusV != HIPSPARSE_STATUS_SUCCESS)
  {
    cout << statusV << endl;
  }
  hipDeviceSynchronize();
  NewPDEfieldV0<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, BV, d_PDEvars); //////
  errSync  = hipGetLastError();
  errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess) 
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
  NewPDEfieldOthers<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, layers, BV, d_PDEvars, d_alt_PDEvars); //////
  cuErrorChecker(errSync, errAsync);

  //hipMemcpy(PDEvars, d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
  //cout << "After second FE step, PDEvars[4305] = " << PDEvars[4305] << endl;

}

void PDE::cuErrorChecker(hipError_t errSync, hipError_t errAsync){
  errSync  = hipGetLastError();
  errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess) 
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}