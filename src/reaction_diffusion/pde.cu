#include "hip/hip_runtime.h"
#include "pde.cpp"
//Allow the CPU implementation for the PDE solver when CUDA is enabled.
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>


#define ARRAY_SIZE 2
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void cuErrorChecker(hipError_t errSync, hipError_t errAsync){
  errSync  = hipGetLastError();
  errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess) 
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}


size_t pbuffersizeH; 
void *pbufferH;
//Needed for cuSparse horizontal and vertical sweeps of ADI
hipsparseStatus_t statusH; 
hipsparseHandle_t handleH;
size_t pbuffersizeV; 
void *pbufferV;
//Needed for cuSparse horizontal and vertical sweeps of ADI
hipsparseStatus_t statusV; 
hipsparseHandle_t handleV;


void PDE::InitialiseCuda(CellularPotts *cpm){
  cout << "Start cuda init" << endl;
  //AllocateTridiagonalvars(sizex, sizey);

  hipMalloc((void**) &d_diffusioncoefficient, sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMalloc((void**) &d_celltype, sizex*sizey*sizeof(int));
  hipMalloc((void**) &d_sigmafield, sizex*sizey*sizeof(int));

  hipMalloc((void**) &d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMemcpy(d_PDEvars, PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);
  hipMalloc((void**) &d_alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMemcpy(d_alt_PDEvars, alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);


  //Needed for ADI steps
  gpuErrchk(hipMallocManaged(&upperH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&diagH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&lowerH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&BH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&upperV, sizey*sizex*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&diagV, sizey*sizex*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&lowerV, sizey*sizex*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&BV, sizey*sizex*sizeof(PDEFIELD_TYPE)));

  handleH = 0;
  pbuffersizeH = 0;
  pbufferH = NULL;
  statusH=hipsparseCreate(&handleH);
  #ifdef PDEFIELD_DOUBLE
    hipsparseDgtsvInterleavedBatch_bufferSizeExt(handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey, &pbuffersizeH); //Compute required buffersize for horizontal sweep
  #else
    hipsparseSgtsvInterleavedBatch_bufferSizeExt(handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey, &pbuffersizeH); //Compute required buffersize for horizontal sweep
  #endif
  gpuErrchk(hipMalloc( &pbufferH, sizeof(char)* pbuffersizeH));
  

  handleV = 0;
  pbuffersizeV = 0;
  pbufferV = NULL;
  statusV=hipsparseCreate(&handleV);
  #ifdef PDEFIELD_DOUBLE
    hipsparseDgtsvInterleavedBatch_bufferSizeExt(handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex, &pbuffersizeV); //Compute required buffersize for vertical sweep
  #else
    hipsparseSgtsvInterleavedBatch_bufferSizeExt(handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex, &pbuffersizeV); //Compute required buffersize for vertical sweep
  #endif
  gpuErrchk(hipMalloc( &pbufferV, sizeof(char)* pbuffersizeV));

  cout << "End cuda init" << endl;
}


__global__ void InitialiseDiagonals(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* lowerH, PDEFIELD_TYPE* upperH, PDEFIELD_TYPE* diagH, PDEFIELD_TYPE* lowerV, PDEFIELD_TYPE* upperV, PDEFIELD_TYPE* diagV, PDEFIELD_TYPE* diffusioncoefficient){
  //This function could in theory be parellelized further, split into 6 (each part only assigning 1 value.), but this is probably slower
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc; //position we currently want to assign to
  int yloc;
  int idcc; //id corresponding to the diffusioncoefficient (+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id/sizey; //needed to obtain interleaved format
    yloc = id%sizey;
    idcc = xloc*sizey + yloc;
    if(xloc == 0){
      lowerH[id] = 0;
      diagH[id] = diffusioncoefficient[idcc+sizey]/dx2 + twooverdt;
      upperH[id] = -diffusioncoefficient[idcc+sizey]/dx2;  
    }
    else if(xloc == sizex -1){
      lowerH[id] = -diffusioncoefficient[idcc-sizey]/dx2;
      diagH[id] = diffusioncoefficient[idcc-sizey]/dx2 + twooverdt;
      upperH[id] = 0;
    }
    else{
      lowerH[id] = -diffusioncoefficient[idcc-sizey]/dx2;
      diagH[id] = (diffusioncoefficient[idcc+sizey]+diffusioncoefficient[idcc-sizey])/dx2 + twooverdt;
      upperH[id] = -diffusioncoefficient[idcc+sizey]/dx2;
    }

    xloc = id%sizex; //needed to obtain interleaved format
    yloc = id/sizex;
    idcc = xloc*sizey + yloc;
    if(yloc == 0){
      lowerV[id] = 0;
      diagV[id] = diffusioncoefficient[idcc+1]/dx2 + twooverdt;
      upperV[id] = -diffusioncoefficient[idcc+1]/dx2;
    }
    else if(yloc == sizey -1){
      lowerV[id] = -diffusioncoefficient[idcc-1]/dx2;
      diagV[id] = diffusioncoefficient[idcc-1]/dx2 + twooverdt;
      upperV[id] = 0;
    }
    else{
      lowerV[id] = -diffusioncoefficient[idcc-1]/dx2;
      diagV[id] = (diffusioncoefficient[idcc+1]+diffusioncoefficient[idcc-1])/dx2 + twooverdt;
      upperV[id] = -diffusioncoefficient[idcc+1]/dx2;
      
    }
  }
}

__global__ void InitialiseHorizontalVectors(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* diffusioncoefficient, PDEFIELD_TYPE* alt_PDEvars){
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc;
  int yloc;
  int idcc; //id corresponding to the diffusioncoefficient and alt_PDEvars(+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id/sizey; //needed to obtain interleaved format
    yloc = id%sizey;
    idcc = xloc*sizey + yloc;

    if (yloc == 0)
      BH[id] = twooverdt*alt_PDEvars[idcc] + (diffusioncoefficient[idcc+1]*(alt_PDEvars[idcc+1] - alt_PDEvars[idcc]))/dx2; 
    else if (yloc == sizey-1)
      BH[id] = twooverdt*alt_PDEvars[idcc] + (diffusioncoefficient[idcc-1]*(alt_PDEvars[idcc-1] - alt_PDEvars[idcc]))/dx2;  
    else 
      BH[id] = twooverdt*alt_PDEvars[idcc] + (diffusioncoefficient[idcc+1]*(alt_PDEvars[idcc+1] - alt_PDEvars[idcc]) + diffusioncoefficient[idcc-1]*(alt_PDEvars[idcc-1] - alt_PDEvars[idcc]))/dx2;
  }
}

__global__ void InitialiseVerticalVectors(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* BV, PDEFIELD_TYPE* diffusioncoefficient, PDEFIELD_TYPE* alt_PDEvars){

  //This function could in theory be parellelized further, split into 6 (each part only assigning 1 value.), but this is probably slower
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc;
  int yloc;
  int idcc; //id corresponding to the diffusioncoefficient and alt_PDEvars(+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id%sizex;
    yloc = id/sizex;
    idcc = xloc*sizey + yloc;

    if (xloc == 0)
      BV[id] = twooverdt*alt_PDEvars[idcc] + (diffusioncoefficient[idcc+sizey]*(alt_PDEvars[idcc+sizey] - alt_PDEvars[idcc]))/dx2; 
    else if (xloc == sizex-1)
      BV[id] = twooverdt*alt_PDEvars[idcc] + (diffusioncoefficient[idcc-sizey]*(alt_PDEvars[idcc-sizey] - alt_PDEvars[idcc]))/dx2;  
    else 
      BV[id] = twooverdt*alt_PDEvars[idcc] + (diffusioncoefficient[idcc+sizey]*(alt_PDEvars[idcc+sizey] - alt_PDEvars[idcc]) + diffusioncoefficient[idcc-sizey]*(alt_PDEvars[idcc-sizey] - alt_PDEvars[idcc]))/dx2;
  }

}



__global__ void NewPDEfieldH0(int sizex, int sizey, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* PDEvars){ //Take the values from BH and assign the new values of the first layers of PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride)
    PDEvars[id] = BH[id];      
}


__global__ void NewPDEfieldV0(int sizex, int sizey, PDEFIELD_TYPE* BV, PDEFIELD_TYPE* PDEvars){ //Take the values from BV and assign the new values of the first layers of PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride){
    PDEvars[sizey*(id%sizex)+id/sizex] = BV[id]; //Conversion is needed because PDEvars iterates over columns first and then rows, while BV does the opposite 
  }
}


__global__ void NewPDEfieldOthers(int sizex, int sizey, int layers, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars){ //copy the other values from alt_PDEvars to PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index+sizex*sizey; id < layers*sizex*sizey; id += stride)
    PDEvars[id] = alt_PDEvars[id]; 
}

__device__ void derivsFitzHughNagumo(PDEFIELD_TYPE current_time, PDEFIELD_TYPE* y, PDEFIELD_TYPE* dydt, bool celltype2, int* sigmafield, int id ){
  
    PDEFIELD_TYPE a = 0.1;
    PDEFIELD_TYPE epsilon = 10;
    PDEFIELD_TYPE beta = -1.0;
    PDEFIELD_TYPE RIext = 0;
    PDEFIELD_TYPE c = 0.191;
    PDEFIELD_TYPE timescale = 40;
  
  dydt[0] = timescale*(epsilon*(y[0]*(1-y[0])*(y[0]-beta)) - y[1] + RIext);
  dydt[1] = timescale*((y[0] -a*y[1]+c));      

}

__global__ void ODEstepFE(PDEFIELD_TYPE dt, PDEFIELD_TYPE ddt, double thetime, int layers, int sizex, int sizey, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars, int* celltype, int* sigmafield){
  
  int nr_of_iterations = round(dt/ddt);
  if (fabs(dt/ddt - nr_of_iterations) > 0.001)
    printf("dt and ddt do not divide properly!");
  PDEFIELD_TYPE begin_time,stepsize_did,stepsize, end_time;
  PDEFIELD_TYPE yscal[ARRAY_SIZE];
  PDEFIELD_TYPE y[ARRAY_SIZE];
  PDEFIELD_TYPE y_new[ARRAY_SIZE];
  PDEFIELD_TYPE dydt[ARRAY_SIZE];
  PDEFIELD_TYPE current_time;
  PDEFIELD_TYPE MaxTimeError = 5e-7;
  PDEFIELD_TYPE stepsize_overshot;
  bool overshot = false;
  bool celltype2 = false;
  int i;

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride){
    if (celltype[id] < 1){
      for (i = 0; i < layers; i++) //fill with current PDE values
        alt_PDEvars[i*sizex*sizey + id]= PDEvars[i*sizex*sizey + id];
    }
    
    else{
      celltype2 = false; 
      if (celltype[id] == 2)
        celltype2 = true;
      begin_time = thetime;
      current_time = thetime;
      end_time = thetime + dt;
      for (i=0;i<layers;i++)
        y[i]=PDEvars[i*sizex*sizey + id];
      for (int it = 0; it < nr_of_iterations; it++){
        derivsFitzHughNagumo(current_time,y,dydt,celltype2, sigmafield,  id);
        current_time += ddt;
        if (it == nr_of_iterations-1) { //Are we done?
          for (i=0;i<layers;i++) {
            alt_PDEvars[i*sizex*sizey + id] = y[i]+ddt*dydt[i];
          }
        }
        else{  
          for (i=0;i<layers;i++) {
            y[i]=y[i]+ddt*dydt[i];  
          }
        }
      }
    }
  }
}



__global__ void CopyAltToOriginalPDEvars(int sizex, int sizey, int layers, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < layers*sizex*sizey; id += stride){
    PDEvars[id] = alt_PDEvars[id]; 
  }
}

__global__ void CopyOriginalToAltPDEvars(int sizex, int sizey, int layers, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < layers*sizex*sizey; id += stride){
    alt_PDEvars[id] = PDEvars[id]; 
  }
}

void PDE::cuPDEsteps(CellularPotts * cpm, int repeat){
  //copy current diffusioncoefficient matrix and celltype matrix from host to device

  hipError_t errSync;
  hipError_t errAsync;
  sigmafield = cpm->getSigma(); 
  hipMemcpy(d_diffusioncoefficient, DiffCoeffs[0][0], sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice); 
  hipMemcpy(d_celltype, celltype[0], sizex*sizey*sizeof(int), hipMemcpyHostToDevice); 
  hipMemcpy(d_sigmafield, sigmafield[0], sizex*sizey*sizeof(int), hipMemcpyHostToDevice); 
  hipMemcpy(d_PDEvars, PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice); 

  
  int nr_blocks = sizex*sizey/par.threads_per_core + 1;
  PDEFIELD_TYPE Cm_maleckar = 50; //in nF
  PDEFIELD_TYPE I_m;
  bool afterdiffusion;

  for (int iteration = 0; iteration < repeat; iteration++){
      //cout << "Iteration = " << iteration << endl;

      //setup matrices for upperdiagonal, diagonal and lower diagonal for both the horizontal and vertical direction, since these remain the same during once MCS
    InitialiseDiagonals<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, lowerH, upperH, diagH, lowerV, upperV, diagV, d_diffusioncoefficient);
    hipDeviceSynchronize();
    errSync  = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
      printf("Async kernel error: %s\n", hipGetErrorString(errAsync));


    cuODEstep();
  
    cuHorizontalADIstep();

    //increase time by dt/2
    thetime = thetime + dt/2;  
    cuODEstep();


    cuVerticalADIstep();
      
    //hipMemcpy(alt_PDEvars, d_alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
    //cout << "After second FE step, alt_PDEvars[23885] = " << alt_PDEvars[23885] << endl;
    

    
    //increase time by dt/2
    thetime = thetime + dt/2; 
 
  }
  hipMemcpy(PDEvars, d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
}

void PDE::cuODEstep(){
  //Do an ODE step of size dt/2
  hipError_t errSync;
  hipError_t errAsync;
  ODEstepFE<<<par.number_of_cores, par.threads_per_core>>>(dt/2, ddt, thetime, layers, sizex, sizey, d_PDEvars, d_alt_PDEvars, d_celltype, d_sigmafield);
  //CopyOriginalToAltPDEvars<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, layers, d_PDEvars, d_alt_PDEvars);

  //hipMemcpy(alt_PDEvars, d_alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
  //cout << "After second FE step, alt_PDEvars[4305] = " << alt_PDEvars[4305] << endl;
  cuErrorChecker(errSync, errAsync);
}

void PDE::cuHorizontalADIstep(){
  //Do a horizontal ADI sweep of size dt/2
  hipError_t errSync;
  hipError_t errAsync;
  InitialiseHorizontalVectors<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, BH, d_diffusioncoefficient, d_alt_PDEvars);
  cuErrorChecker(errSync, errAsync);
  #ifdef PDEFIELD_DOUBLE
    statusH = hipsparseDgtsvInterleavedBatch(handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey, pbufferH);
  #else
    statusH = hipsparseSgtsvInterleavedBatch(handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey, pbufferH);
  #endif
  if (statusH != HIPSPARSE_STATUS_SUCCESS)
  {
    cout << statusH << endl;
  }
  NewPDEfieldH0<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, BH, d_PDEvars);    
  cuErrorChecker(errSync, errAsync);
  NewPDEfieldOthers<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, layers, BV, d_PDEvars, d_alt_PDEvars); //////
  cuErrorChecker(errSync, errAsync);

  //hipMemcpy(PDEvars, d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
  //cout << "After second FE step, PDEvars[4305] = " << PDEvars[4305] << endl;

}

void PDE::cuVerticalADIstep(){
  //Do a vertical ADI sweep of size dt/2
  hipError_t errSync;
  hipError_t errAsync;
  InitialiseVerticalVectors<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, BV, d_diffusioncoefficient, d_alt_PDEvars);
  cuErrorChecker(errSync, errAsync);
  #ifdef PDEFIELD_DOUBLE
    statusV = hipsparseDgtsvInterleavedBatch(handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex, pbufferV);
  #else
    statusV = hipsparseSgtsvInterleavedBatch(handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex, pbufferV);
  #endif
  if (statusV != HIPSPARSE_STATUS_SUCCESS)
  {
    cout << statusV << endl;
  }
  hipDeviceSynchronize();
  NewPDEfieldV0<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, BV, d_PDEvars); //////
  errSync  = hipGetLastError();
  errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess) 
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
  NewPDEfieldOthers<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, layers, BV, d_PDEvars, d_alt_PDEvars); //////
  cuErrorChecker(errSync, errAsync);

  //hipMemcpy(PDEvars, d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
  //cout << "After second FE step, PDEvars[4305] = " << PDEvars[4305] << endl;

}

